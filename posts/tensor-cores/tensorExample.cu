#include <iostream>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <string>
#include <nvml.h>

#define WMMA_TILE   16  // WMMA supports 16x16 tiles
// #define DEBUG 0

using namespace nvcuda;

void checkNvmlError(nvmlReturn_t result, const char* msg) {
    if (result != NVML_SUCCESS) {
        std::cerr << "Error: " << msg << " - " << nvmlErrorString(result) << std::endl;
        exit(1);
    }
}

template <typename T>
void printMatrix(T* matrix, int N, int M) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < M; ++j) {
            printf("%2.0f ", float(matrix[i * M + j]));
        }
        printf("\n");
    }
    printf("\n");
}

__device__ void loadSharedMemory(half* A, half* B, float* C, half* sharedA, half* sharedB, float* sharedC, int N, int M, int tileRow, int tileCol, int sharedTile, int TILE_BLOCKS) {
    // * tile at the warp level for loading shared memory
    for (int wmmaTile = 0; wmmaTile < TILE_BLOCKS * WMMA_TILE; wmmaTile+=WMMA_TILE) {
        // * these values can get values between 0 - TILE_BLOCK * WMMA_TILE
        // * e.g., when TILE_BLOCK is equal to 2, these values can get 0 to 31

        // * Compute the indices for loading the data inside the shared memories
        int sharedAxId = threadIdx.x * TILE_BLOCKS * WMMA_TILE;
        int sharedAyId = threadIdx.y + wmmaTile;
        int sharedAId = sharedAxId + sharedAyId;

        int sharedBxId = (threadIdx.x + wmmaTile) * WMMA_TILE;
        int sharedById = threadIdx.y;
        int sharedBId = sharedBxId + sharedById;

        // * Compute the indices for reading the data from the global memory
        int globalAx = (tileRow + threadIdx.x) * N;
        int globalAy = sharedTile + wmmaTile + threadIdx.y;
        int globalA = globalAx + globalAy;

        int globalBx = (threadIdx.x + sharedTile + wmmaTile) * N;
        int globalBy = tileCol + threadIdx.y;
        int globalB = globalBx + globalBy;

        // * Load the input values from global memory to shared memory
        sharedA[sharedAId] = A[globalA];
        sharedB[sharedBId] = B[globalB];

        // * Since C must be only loaded once
        if (sharedTile == 0 && wmmaTile == 0) {
            int sharedCxId = threadIdx.x * WMMA_TILE;
            int sharedCyId = threadIdx.y;
            int sharedCId = sharedCxId + sharedCyId;

            int globalCx = (tileRow + threadIdx.x) * N;
            int globalCy = tileCol + threadIdx.y;
            int globalC = globalCx + globalCy;

            sharedC[sharedCId] = C[globalC];
        }
    }
}

__device__ void storeGlobalMemory(float* D, float* sharedC, int N, int tileRow, int tileCol) {
    int globalCxStore = (tileRow + threadIdx.x) * N;
    int globalCyStore = tileCol + threadIdx.y;
    int globalCStore = globalCxStore + globalCyStore;

    int cxStore = threadIdx.x * WMMA_TILE;
    int cyStore = threadIdx.y;
    int cStore = cxStore + cyStore;

    D[globalCStore] = sharedC[cStore];
}

__device__ void updateBFrag(
    wmma::fragment<wmma::matrix_b, WMMA_TILE, WMMA_TILE, WMMA_TILE, half, wmma::row_major> b_frag_0, 
    wmma::fragment<wmma::matrix_b, WMMA_TILE, WMMA_TILE, WMMA_TILE, half, wmma::row_major> b_frag_1) {
        // * SAFE MODE TCU 0 <- TCU 1
        for(int i=0; i < b_frag_0.num_elements / 4; i++) {
            b_frag_0.x[i] = b_frag_0.x[i + 4];
        }

        // * SAFE MODE TCU 1 <- TCU 0
        for(int i=0; i < b_frag_1.num_elements / 4; i++) {
            b_frag_1.x[i + 4] = b_frag_1.x[i];
        }
}

__device__ void checkCFrag(
    wmma::fragment<wmma::accumulator, WMMA_TILE, WMMA_TILE, WMMA_TILE, float> c_frag_0, 
    wmma::fragment<wmma::accumulator, WMMA_TILE, WMMA_TILE, WMMA_TILE, float> c_frag_1,
    int *fault) {
        for(int i=0; i < 4; i++) {
            if (c_frag_0.x[i] != c_frag_0.x[i + 4]) {
                printf("Something happened frag0!!!\n");
                fault[0] = -1;
            }
            if (c_frag_1.x[i] != c_frag_1.x[i + 4]) {
                printf("Something happened frag1!!!\n");
                fault[0] = 1;
            }
        }
}

__device__ int wmma_diagnosis(
   wmma::fragment<wmma::matrix_a, WMMA_TILE, WMMA_TILE, WMMA_TILE, half, wmma::row_major> fragA,
   wmma::fragment<wmma::matrix_b, WMMA_TILE, WMMA_TILE, WMMA_TILE, half, wmma::row_major> fragB,
   const float*  fragC,
   int N, int M) {

    int laneid;
    asm("mov.u32 %0, %laneid;" :"=r"(laneid));
    int bCol = (int)(laneid / 4);
    int bRow = (int)(laneid % 4);

    // Declare the fragments
    wmma::fragment<wmma::matrix_b, WMMA_TILE, WMMA_TILE, WMMA_TILE, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_TILE, WMMA_TILE, WMMA_TILE, float> acc_frag;
    wmma::fill_fragment(acc_frag, 0.0f);

    __shared__ half diagnosis[256];
    __shared__ float Cdiagnosis[256];

    // * fill the b diagnosis within the fragment data
    for (int i = 0; i < 2; i++) {
        diagnosis[((bRow * 2) + i) * WMMA_TILE + bCol] = fragB.x[i];
        diagnosis[((bRow * 2) + i  + 8) * WMMA_TILE + bCol] = fragB.x[i + 2];

        diagnosis[((bRow * 2) + i) * WMMA_TILE + bCol + 8] = fragB.x[i + 4];
        diagnosis[((bRow * 2) + i  + 8) * WMMA_TILE + bCol + 8] = fragB.x[i + 6];
    }

    __syncthreads();

    // * Copy the columns into the following ones
    if (bCol < 4) {
        for (int i = 0; i < 16; i++) {
            diagnosis[i * WMMA_TILE + bCol + 4] = diagnosis[i * WMMA_TILE + bCol];
            diagnosis[i * WMMA_TILE + bCol + 12] = diagnosis[i * WMMA_TILE + bCol + 8];
        }
    }

    wmma::load_matrix_sync(b_frag, diagnosis, WMMA_TILE);
    wmma::mma_sync(acc_frag, fragA, b_frag, acc_frag);

    __syncthreads();

    // * identification
    int cRow = (int)(laneid / 4);
    int cCol = (int)(laneid % 4);

    // * fill the diagnosis matrix with the c fragment data
    for (int i = 0; i < 2; i++) {
        Cdiagnosis[((cRow) * WMMA_TILE) + (cCol * 2) + i] = acc_frag.x[i];
        Cdiagnosis[((cRow + 8) * WMMA_TILE) + (cCol * 2) + i] = acc_frag.x[i + 2];
        Cdiagnosis[((cRow) * WMMA_TILE) + (cCol * 2) + 8 + i] = acc_frag.x[i + 4];
        Cdiagnosis[((cRow + 8) * WMMA_TILE) + (cCol * 2) + 8 + i] = acc_frag.x[i + 6];
    }

    __syncthreads();

    // * diagnosis
    if (laneid == 0) {
        for (int i = 0; i < 16; i++) {
            for (int j = 0; j < 4; j++) {
                if (Cdiagnosis[i * WMMA_TILE + j] != Cdiagnosis[i * WMMA_TILE + j + 4]) {
                    // * faulty TCU0
                    return 0;
                }
                if (Cdiagnosis[i * WMMA_TILE + j + 8] != Cdiagnosis[i * WMMA_TILE + j + 12]) { 
                    // * faulty TCU1
                    return 1;
                }
            }
        }
    }
    // * the fault has not been detected
    return -1;
}


__global__ void matrixMulAddWMMACorrection(half* A, half* B, float* C, float* D, int N, int M, int *fault, int* tcu, int TILE_BLOCKS) {
    extern __shared__ half sharedMem[];
    half* sharedA = sharedMem;
    half* sharedB = sharedMem + TILE_BLOCKS * WMMA_TILE * WMMA_TILE;
    float* sharedC = (float*)(sharedMem + 2 * TILE_BLOCKS * WMMA_TILE * WMMA_TILE);

    int tileRow = blockIdx.y * WMMA_TILE;
    int tileCol = blockIdx.x * WMMA_TILE;

    wmma::fragment<wmma::matrix_a, WMMA_TILE, WMMA_TILE, WMMA_TILE, half, wmma::row_major> a_frag;

    wmma::fragment<wmma::matrix_b, WMMA_TILE, WMMA_TILE, WMMA_TILE, half, wmma::row_major> b_frag_0; 
    wmma::fragment<wmma::matrix_b, WMMA_TILE, WMMA_TILE, WMMA_TILE, half, wmma::row_major> b_frag_1;
    wmma::fragment<wmma::accumulator, WMMA_TILE, WMMA_TILE, WMMA_TILE, float> acc_frag_0;
    wmma::fragment<wmma::accumulator, WMMA_TILE, WMMA_TILE, WMMA_TILE, float> acc_frag_1;
    wmma::fragment<wmma::accumulator, WMMA_TILE, WMMA_TILE, WMMA_TILE, float> c_frag_0;
    wmma::fragment<wmma::accumulator, WMMA_TILE, WMMA_TILE, WMMA_TILE, float> c_frag_1;

    wmma::fill_fragment(acc_frag_0, 0.0f);
    wmma::fill_fragment(acc_frag_1, 0.0f);

    // if (tileRow == 0 && tileCol == 0) {
    // * validate that the indices are inside the matrices
    if (tileRow < M && tileCol < N) {

        // * tile at the device level
        for (int sharedTile = 0; sharedTile < M; sharedTile += TILE_BLOCKS * WMMA_TILE) {

            loadSharedMemory(A, B, C, sharedA, sharedB, sharedC, N, M, tileRow, tileCol, sharedTile, TILE_BLOCKS);

            __syncthreads();

            // * tile at the warp level for performing matrix multiplciation A * B for each segment
            for (int wmmaTile = 0; wmmaTile < TILE_BLOCKS * WMMA_TILE; wmmaTile+=WMMA_TILE) {
                wmma::load_matrix_sync(a_frag, sharedA + wmmaTile, WMMA_TILE);

                wmma::load_matrix_sync(b_frag_0, sharedB, WMMA_TILE);
                wmma::load_matrix_sync(b_frag_1, sharedB, WMMA_TILE);

                updateBFrag(b_frag_0, b_frag_1);

                wmma::mma_sync(acc_frag_0, a_frag, b_frag_0, acc_frag_0);
                wmma::mma_sync(acc_frag_1, a_frag, b_frag_1, acc_frag_1);
            }
        }
        // * add the C segment
        wmma::load_matrix_sync(c_frag_0, sharedC, WMMA_TILE, wmma::mem_row_major);
        wmma::load_matrix_sync(c_frag_1, sharedC, WMMA_TILE, wmma::mem_row_major);

#pragma unroll
        for(int i=0; i < c_frag_0.num_elements; i++) {
            c_frag_0.x[i] = acc_frag_0.x[i] + c_frag_0.x[i];
        }
#pragma unroll
        for(int i=0; i < c_frag_1.num_elements; i++) {
            c_frag_1.x[i] = acc_frag_1.x[i] + c_frag_1.x[i];
        }

        __syncthreads();

        checkCFrag(c_frag_0, c_frag_1, fault);

        __syncthreads();

        if (fault[0] == -1) {
            tcu[0] = wmma_diagnosis(a_frag, b_frag_0, sharedC, N, M);
        } else if  (fault[0] == 1) {
            tcu[0] = wmma_diagnosis(a_frag, b_frag_1, sharedC, N, M);
        }

        // * store the output segment from the fragment into the shared memory
        __syncthreads();

        if (tcu[0] == 0){
            // * faulty TCU 0, means consider only the TCU 1 data
            for(int i=0; i < 4; i++) {
                c_frag_0.x[i] = c_frag_1.x[i + 4];
            }
            wmma::store_matrix_sync(sharedC, c_frag_0, WMMA_TILE, wmma::mem_row_major);
        } else {
            // * faulty TCU 1, means consider only the TCU 0 data
            for(int i=0; i < 4; i++) {
                c_frag_1.x[i + 4] = c_frag_0.x[i];
            }
            wmma::store_matrix_sync(sharedC, c_frag_1, WMMA_TILE, wmma::mem_row_major);
        }

        storeGlobalMemory(D, sharedC, N, tileRow, tileCol);
    }
}

__global__ void matrixMulAddWMMADetection(half* A, half* B, float* C, float* D, int N, int M, int *fault, int TILE_BLOCKS) {
    extern __shared__ half sharedMem[];
    half* sharedA = sharedMem;
    half* sharedB = sharedMem + TILE_BLOCKS * WMMA_TILE * WMMA_TILE;
    float* sharedC = (float*)(sharedMem + 2 * TILE_BLOCKS * WMMA_TILE * WMMA_TILE);

    int tileRow = blockIdx.y * WMMA_TILE;
    int tileCol = blockIdx.x * WMMA_TILE;

    wmma::fragment<wmma::matrix_a, WMMA_TILE, WMMA_TILE, WMMA_TILE, half, wmma::row_major> a_frag;

    wmma::fragment<wmma::matrix_b, WMMA_TILE, WMMA_TILE, WMMA_TILE, half, wmma::row_major> b_frag_0; 
    wmma::fragment<wmma::matrix_b, WMMA_TILE, WMMA_TILE, WMMA_TILE, half, wmma::row_major> b_frag_1;
    wmma::fragment<wmma::accumulator, WMMA_TILE, WMMA_TILE, WMMA_TILE, float> acc_frag_0;
    wmma::fragment<wmma::accumulator, WMMA_TILE, WMMA_TILE, WMMA_TILE, float> acc_frag_1;
    wmma::fragment<wmma::accumulator, WMMA_TILE, WMMA_TILE, WMMA_TILE, float> c_frag_0;
    wmma::fragment<wmma::accumulator, WMMA_TILE, WMMA_TILE, WMMA_TILE, float> c_frag_1;

    wmma::fill_fragment(acc_frag_0, 0.0f);
    wmma::fill_fragment(acc_frag_1, 0.0f);

    // if (tileRow == 0 && tileCol == 0) {
    // * validate that the indices are inside the matrices
    if (tileRow < M && tileCol < N) {

        // * tile at the device level
        for (int sharedTile = 0; sharedTile < M; sharedTile += TILE_BLOCKS * WMMA_TILE) {

            loadSharedMemory(A, B, C, sharedA, sharedB, sharedC, N, M, tileRow, tileCol, sharedTile, TILE_BLOCKS);

            __syncthreads();

            // * tile at the warp level for performing matrix multiplciation A * B for each segment
            for (int wmmaTile = 0; wmmaTile < TILE_BLOCKS * WMMA_TILE; wmmaTile+=WMMA_TILE) {
                wmma::load_matrix_sync(a_frag, sharedA + wmmaTile, WMMA_TILE);

                wmma::load_matrix_sync(b_frag_0, sharedB, WMMA_TILE);
                wmma::load_matrix_sync(b_frag_1, sharedB, WMMA_TILE);

                updateBFrag(b_frag_0, b_frag_1);

                wmma::mma_sync(acc_frag_0, a_frag, b_frag_0, acc_frag_0);
                wmma::mma_sync(acc_frag_1, a_frag, b_frag_1, acc_frag_1);
            }
        }
        // * add the C segment
        wmma::load_matrix_sync(c_frag_0, sharedC, WMMA_TILE, wmma::mem_row_major);
        wmma::load_matrix_sync(c_frag_1, sharedC, WMMA_TILE, wmma::mem_row_major);

#pragma unroll
        for(int i=0; i < c_frag_0.num_elements; i++) {
            c_frag_0.x[i] = acc_frag_0.x[i] + c_frag_0.x[i];
        }
#pragma unroll
        for(int i=0; i < c_frag_1.num_elements; i++) {
            c_frag_1.x[i] = acc_frag_1.x[i] + c_frag_1.x[i];
        }

        __syncthreads();

        checkCFrag(c_frag_0, c_frag_1, fault);

        // * store the output segment from the fragment into the shared memory
        wmma::store_matrix_sync(sharedC, c_frag_0, WMMA_TILE, wmma::mem_row_major);

        storeGlobalMemory(D, sharedC, N, tileRow, tileCol);
    }
}

__global__ void matrixMulAddWMMA(half* A, half* B, float* C, float* D, int N, int M, int TILE_BLOCKS) {
    extern __shared__ half sharedMem[];
    half* sharedA = sharedMem;
    half* sharedB = sharedMem + TILE_BLOCKS * WMMA_TILE * WMMA_TILE;
    float* sharedC = (float*)(sharedMem + 2 * TILE_BLOCKS * WMMA_TILE * WMMA_TILE);

    int tileRow = blockIdx.y * WMMA_TILE;
    int tileCol = blockIdx.x * WMMA_TILE;

    wmma::fragment<wmma::matrix_a, WMMA_TILE, WMMA_TILE, WMMA_TILE, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_TILE, WMMA_TILE, WMMA_TILE, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_TILE, WMMA_TILE, WMMA_TILE, float> acc_frag;
    wmma::fragment<wmma::accumulator, WMMA_TILE, WMMA_TILE, WMMA_TILE, float> c_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    // if (tileRow == 0 && tileCol == 0) {
    // * validate that the indices are inside the matrices
    if (tileRow < M && tileCol < N) {

        // * tile at the device level
        for (int sharedTile = 0; sharedTile < M; sharedTile += TILE_BLOCKS * WMMA_TILE) {

            loadSharedMemory(A, B, C, sharedA, sharedB, sharedC, N, M, tileRow, tileCol, sharedTile, TILE_BLOCKS);

            __syncthreads();

            // * tile at the warp level for performing matrix multiplciation A * B for each segment
            for (int wmmaTile = 0; wmmaTile < TILE_BLOCKS * WMMA_TILE; wmmaTile+=WMMA_TILE) {
                wmma::load_matrix_sync(a_frag, sharedA + wmmaTile, WMMA_TILE);
                wmma::load_matrix_sync(b_frag, sharedB + wmmaTile, WMMA_TILE);

                wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
            }

        }
        // * add the C segment
        wmma::load_matrix_sync(c_frag, sharedC, WMMA_TILE, wmma::mem_row_major);

#pragma unroll
        for(int i=0; i < c_frag.num_elements; i++) {
            c_frag.x[i] = acc_frag.x[i] + c_frag.x[i];
        }

        __syncthreads();

        // * store the output segment from the fragment into the shared memory
        wmma::store_matrix_sync(sharedC, c_frag, WMMA_TILE, wmma::mem_row_major);

        storeGlobalMemory(D, sharedC, N, tileRow, tileCol);
    }
}

// Timing wrapper function
template <typename Func>
float measureKernelTime(Func kernel) {
    hipEvent_t start, stop;
    float elapsedTime;

    // Create CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);

    // Launch the kernel (passed as a lambda)
    kernel();

    // Record the stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;  // Return time in milliseconds
}

void printfStatistics(int method, int N, float timeMs, int sharedMemory, float power, float powerBefore, int temperatureBefore, int temperatureAfter, int clockBefore, int clockAfter, nvmlPstates_t pStateAfter) {
    // * Calculate TFLOPs
    double flops = 2.0 * N * N * N + N * N;
    double elapsedTimeInSeconds = timeMs / 1000.0;
    double tflops = flops / (elapsedTimeInSeconds * 1e12);

    // * method, size, time, shared, flops, tflops
    // * method 0 - normal, 1 - detection, 2 - correction
    if (method == 0) printf("normal,%d,%2.4f,%d,%2.4f,%2.4f,%2.4f,%2.4f,%d,%d,%d,%d,%d\n", N, timeMs, sharedMemory, flops, tflops, powerBefore, power, temperatureBefore, temperatureAfter, clockBefore, clockAfter, (unsigned int)pStateAfter);
    if (method == 1) printf("detection,%d,%2.4f,%d,%2.4f,%2.4f,%2.4f,%2.4f,%d,%d,%d,%d,%d\n", N, timeMs, sharedMemory, flops, tflops, powerBefore, power, temperatureBefore, temperatureAfter, clockBefore, clockAfter, (unsigned int)pStateAfter);
    if (method == 2) printf("correction,%d,%2.4f,%d,%2.4f,%2.4f,%2.4f,%2.4f,%d,%d,%d,%d,%d\n", N, timeMs, sharedMemory, flops, tflops, powerBefore, power, temperatureBefore, temperatureAfter, clockBefore, clockAfter, (unsigned int)pStateAfter);
    
}

int main(int argc, char* argv[]) {
    // * Must be multiples of 16 for wmma code to work
    if (argv[1] == "") {
        printf("Matrix size must be initialized, and it must be multiples of 16");
        return 0;
    }

    // Initialize NVML
    nvmlReturn_t result = nvmlInit();
    checkNvmlError(result, "Failed to initialize NVML");

    // Get the number of GPUs
    unsigned int deviceCount;
    result = nvmlDeviceGetCount(&deviceCount);
    checkNvmlError(result, "Failed to get device count");

    // Select the first GPU (you can modify this to target a different GPU)
    nvmlDevice_t device;
    result = nvmlDeviceGetHandleByIndex(0, &device);
    checkNvmlError(result, "Failed to get handle for device");

    const int M = atoi(argv[1]);
    const int N = atoi(argv[1]);
    const int TILE_BLOCKS = atoi(argv[2]);

    size_t half_bytes = N * M * sizeof(half);
    size_t float_bytes = N * M * sizeof(float);

    int *fault;
    int *fault_device;

    int *tcu;
    int *tcu_device;

    fault = (int*)malloc(sizeof(int));
    fault[0] = 0;
    hipMalloc((void**)&fault_device, sizeof(int));

    tcu = (int*)malloc(sizeof(int));
    tcu[0] = -1;
    hipMalloc((void**)&tcu_device, sizeof(int));

    half* h_A = (half*)malloc(half_bytes);
    half* h_B = (half*)malloc(half_bytes);
    float* h_C = (float*)malloc(float_bytes);
    float* h_D = (float*)malloc(float_bytes);

    for (int i = 0; i < N * M; i++) {
        h_A[i] = __float2half(static_cast<float>(1));
        h_B[i] = __float2half(static_cast<float>(1));
        h_C[i] = static_cast<float>(1);
    }

    half* d_A, * d_B;
    float* d_C, * d_D;
    hipMalloc(&d_A, half_bytes);
    hipMalloc(&d_B, half_bytes);
    hipMalloc(&d_C, float_bytes);
    hipMalloc(&d_D, float_bytes);

    hipMemcpy(d_A, h_A, half_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, half_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, float_bytes, hipMemcpyHostToDevice);

    dim3 blockDim(WMMA_TILE, WMMA_TILE);
    dim3 gridDim(N / WMMA_TILE, M / WMMA_TILE);

    size_t sharedMemSize = 2 * TILE_BLOCKS * WMMA_TILE * WMMA_TILE * sizeof(half) + WMMA_TILE * WMMA_TILE * sizeof(float);

    // Record power usage before kernel execution
    unsigned int powerBefore;
    result = nvmlDeviceGetPowerUsage(device, &powerBefore);
    checkNvmlError(result, "Failed to get power usage");

    // Get temperature before kernel execution
    unsigned int temperatureBefore;
    result = nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperatureBefore);
    checkNvmlError(result, "Failed to get temperature");

    // Get GPU clock frequency before kernel execution
    unsigned int clockBefore;
    result = nvmlDeviceGetClockInfo(device, NVML_CLOCK_GRAPHICS, &clockBefore);
    checkNvmlError(result, "Failed to get GPU clock frequency");
    
    // Get P-state (performance state) before kernel execution
    nvmlPstates_t pStateBefore;
    result = nvmlDeviceGetPerformanceState(device, &pStateBefore);
    checkNvmlError(result, "Failed to get performance state");

    // * Measure kernel execution time using the wrapper
    float timeMs = measureKernelTime([&]() {
        matrixMulAddWMMADetection<<<gridDim, blockDim, sharedMemSize>>>(d_A, d_B, d_C, d_D, N, M, fault_device, TILE_BLOCKS);
    });

    // Record power usage after kernel execution
    unsigned int powerAfter;
    result = nvmlDeviceGetPowerUsage(device, &powerAfter);
    checkNvmlError(result, "Failed to get power usage");

    // Get temperature after kernel execution
    unsigned int temperatureAfter;
    result = nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperatureAfter);
    checkNvmlError(result, "Failed to get temperature");

    // Get GPU clock frequency after kernel execution
    unsigned int clockAfter;
    result = nvmlDeviceGetClockInfo(device, NVML_CLOCK_GRAPHICS, &clockAfter);
    checkNvmlError(result, "Failed to get GPU clock frequency");

    // Get P-state (performance state) after kernel execution
    nvmlPstates_t pStateAfter;
    result = nvmlDeviceGetPerformanceState(device, &pStateAfter);
    checkNvmlError(result, "Failed to get performance state");

    printfStatistics(1, N, timeMs, sharedMemSize, powerAfter / 1000.0, powerBefore / 1000.0, temperatureBefore, temperatureAfter, clockBefore, clockAfter, pStateAfter);

    // * Measure kernel execution time using the wrapper
    timeMs = measureKernelTime([&]() {
        matrixMulAddWMMACorrection<<<gridDim, blockDim, sharedMemSize>>>(d_A, d_B, d_C, d_D, N, M, fault_device, tcu_device, TILE_BLOCKS);
    });

    result = nvmlDeviceGetPowerUsage(device, &powerAfter);
    checkNvmlError(result, "Failed to get power usage");

    // Get temperature after kernel execution
    result = nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperatureAfter);
    checkNvmlError(result, "Failed to get temperature");

    // Get GPU clock frequency after kernel execution
    result = nvmlDeviceGetClockInfo(device, NVML_CLOCK_GRAPHICS, &clockAfter);
    checkNvmlError(result, "Failed to get GPU clock frequency");

    // Get P-state (performance state) after kernel execution
    result = nvmlDeviceGetPerformanceState(device, &pStateAfter);
    checkNvmlError(result, "Failed to get performance state");

    printfStatistics(2, N, timeMs, sharedMemSize, powerAfter / 1000.0, powerBefore / 1000.0, temperatureBefore, temperatureAfter, clockBefore, clockAfter, pStateAfter);

    // * Measure kernel execution time using the wrapper
    timeMs = measureKernelTime([&]() {
        matrixMulAddWMMA<<<gridDim, blockDim, sharedMemSize>>>(d_A, d_B, d_C, d_D, N, M, TILE_BLOCKS);
    });

    result = nvmlDeviceGetPowerUsage(device, &powerAfter);
    checkNvmlError(result, "Failed to get power usage");

    // Get temperature after kernel execution
    result = nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperatureAfter);
    checkNvmlError(result, "Failed to get temperature");

    // Get GPU clock frequency after kernel execution
    result = nvmlDeviceGetClockInfo(device, NVML_CLOCK_GRAPHICS, &clockAfter);
    checkNvmlError(result, "Failed to get GPU clock frequency");

    // Get P-state (performance state) after kernel execution
    result = nvmlDeviceGetPerformanceState(device, &pStateAfter);
    checkNvmlError(result, "Failed to get performance state");

    printfStatistics(0, N, timeMs, sharedMemSize, powerAfter / 1000.0, powerBefore / 1000.0, temperatureBefore, temperatureAfter, clockBefore, clockAfter, pStateAfter);

    hipMemcpy(h_D, d_D, float_bytes, hipMemcpyDeviceToHost);

    hipMemcpy(fault, fault_device, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(tcu, tcu_device, sizeof(int), hipMemcpyDeviceToHost);

    if (fault[0] != 0) {
      printf("Fault detected at the TCU %d\n", tcu[0]);
    }

#ifdef DEBUG
    std::cout << "Matrix A:" << std::endl;
    printMatrix(h_A, N, M);

    std::cout << "Matrix B:" << std::endl;
    printMatrix(h_B, N, M);

    std::cout << "Matrix C:" << std::endl;
    printMatrix(h_C, N, M);

    std::cout << "Matrix D (Result A*B+C):" << std::endl;
    printMatrix(h_D, N, M);
#endif

    // Shutdown NVML
    result = nvmlShutdown();
    checkNvmlError(result, "Failed to shutdown NVML");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    hipFree(tcu_device);
    hipFree(fault_device);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);
    free(tcu);
    free(fault);

    return 0;
}
